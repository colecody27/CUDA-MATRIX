#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#define N 16

//Compute number of even values in a 16 x 16 array 
__global__ void countEvens(int da[N][N], int *dcount){
    //Find location of thread
    int x = threadIdx.x;
    int y = threadIdx.y;

    //Check if value is even  
    if(da[x][y] % 2 == 0){
        atomicAdd(dcount, 1);
    }         
}

//Compute matrix square
__global__ void computeSquare(int da[N][N], int dsquare[N][N]){
    //Find location of thread
    int row = threadIdx.x;
    int col = threadIdx.y;

    //Compute square
    for(int i = 0; i < N; i++){
            dsquare[row][col] += da[row][i] * da[i][col];
    }
}

int main(){
    /*EXERCISE 1*/
    //Create 2d array with random values
    int arr[N][N];
    for(int i = 0; i < N; i++){
        for(int j = 0; j <  N; j++){
            arr[i][j] = rand() % 30;
        }
    }

    //Print out array
    std::cout << "Random array: \n";
    for(int i = 0; i < N; i++){
        std::cout << "[";
        for(int j = 0; j < N; j++){
            if(j == N-1)
               std::cout << arr[i][j];    
            else
               std::cout << arr[i][j] <<  ", ";
        }
        std::cout << "]\n";
    }

    //Allocate memory on GPU and declare variables
    int *da;
    int *dCount; 
    int hCount;
    hipMalloc((void **)&da, N*N*sizeof(int));
    hipMalloc((void **)&dCount, sizeof(int));

    //Copy array from CPU TO GPU 
    hipMemcpy(da, arr, N*N*sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, N);
    countEvens<<<1, threadsPerBlock>>>((int(*) [N])da, dCount); 

    //Move value from GPU to CPU
    hipMemcpy(&hCount, dCount, sizeof(int), hipMemcpyDeviceToHost);

    //Free device memory 
    hipFree(da);
    hipFree(dCount); 

    std::cout <<  "\nNumber of even values in array is: " << hCount << "\n\n"; 


    /*EXERCISE 2*/
    //Allocate memory on GPU and declare variables 
    int *dsquared;
    int squared[N][N];
    hipMalloc((void **)&da, N*N*sizeof(int));
    hipMalloc((void **)&dsquared, N*N*sizeof(int));

    //Copy array from CPU TO GPU 
    hipMemcpy(da, arr, N*N*sizeof(int), hipMemcpyHostToDevice);

    computeSquare<<<1, threadsPerBlock>>>((int(*) [N])da, (int(*) [N])dsquared); 

    //Move value from GPU to CPU
    hipMemcpy(squared, dsquared, N*N*sizeof(int), hipMemcpyDeviceToHost);

    //Free device memory 
    hipFree(da);
    hipFree(dsquared);

    std::cout << "Squared array is: \n";
    //Print out squared array
    for(int i = 0; i < N; i++){
        std::cout << "[";
        for(int j = 0; j < N; j++){
            if(j == N-1)
               std::cout << squared[i][j];    
            else
               std::cout << squared[i][j] <<  ", ";
        }
        std::cout << "]\n";
    }
    
    return 0; 
}